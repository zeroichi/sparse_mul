#include "hip/hip_runtime.h"
// -*- mode: c++ -*-

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "crs.h"
#include "vec.h"
// this macro requests calling hipDeviceSynchronize() before gettimeofday()
#define MISC_DEVICE_SYNC
#include "misc.h"

__global__ void collect( int *row_ptr, int *tmp, int *result, int rows ) {
    int total = gridDim.x * gridDim.y * blockDim.x; // the number of total threads
    int tid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x; // thread id
    int i, j, start, end;
    for( i=tid; i<rows; i+=total ) {
        start = row_ptr[i];
        end   = row_ptr[i+1];
        result[i] = 0;
        for( j=start; j<end; ++j )
            result[i] += tmp[j];
    }
}

#ifdef WITH_GATHER

__global__ void gather( int *v, int *col_ind, int *gather, int len ) {
    int total = gridDim.x * gridDim.y * blockDim.x; // the number of total threads
    int tid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x; // thread id
    int i;
    for( i=tid; i<len; i+=total )
        gather[i] = v[col_ind[i]];
}

__global__ void compute( int *val, int *gather, int *tmp, int len ) {
    int total = gridDim.x * gridDim.y * blockDim.x; // the number of total threads
    int tid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x; // thread id
    int i;
    for( i=tid; i<len; i+=total )
        tmp[i] = val[i] * gather[i];
}

#else

__global__ void compute( int *val, int *col_ind, int *v, int *tmp, int len ) {
    int total = gridDim.x * gridDim.y * blockDim.x; // the number of total threads
    int tid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x; // thread id
    int i;
    for( i=tid; i<len; i+=total )
        tmp[i] = val[i] * v[col_ind[i]];
}

#endif

int main( int argc, char **argv ) {
	if( argc < 3 ) {
		printf( "usage: %s <matrix file> <vector file>\n", argv[0] );
		return 0;
	}
    crs_t crs;
    vec_t v;
    int r;
    TIMEINIT;
    
    TIMESTART;
	if( strcmp_suffix( argv[1], ".bin" ) == 0 )
        r = crs_load_bin( &crs, argv[1] );
    else
        r = crs_load( &crs, argv[1] );
    if( r != 0 ) {
        printf( "error: failed to load crs file, return code = %d\n", r );
        return r;
    }
    TIMEEND( "loading sparse matrix data" );

    TIMESTART;
    r = vec_load( &v, argv[2] );
    if( r != 0 ) {
        printf( "error: failed to load vector file, return code = %d\n", r );
        return r;
    }
    TIMEEND( "loading dense vector data" );

    if( crs.cols != (int)v.size ) {
		printf( "error: vector size is different with matrix column size\n" );
        crs_delete( &crs );
		vec_delete( &v );
		return 1;
	}

    printf( "matrix filename ... %s\n"
            "matrix size ....... %d x %d\n"
            "matrix nonzeros ... %d\n"
            "vector filename ... %s\n"
            "vector size ....... %d\n"
            "compute device .... gpu\n"
            "with gather ....... %s\n"
            , argv[1]
            , crs.rows, crs.cols
            , crs.nonzeros
            , argv[2]
            , (int)v.size
#ifdef WITH_GATHER
            , "true"
#else
            , "false"
#endif
            );

    struct {
        int *val;     // non-zero values
        int *col_ind; // column index
        int *row_ptr; // row pointer
        int *vector;
        int *tmp;
        int *result;
#ifdef WITH_GATHER
        int *gather;
#endif
    } device;

    size_t alloc_total = 0;
    hipMalloc( (void**)&device.val, sizeof(*device.val) * crs.nonzeros );
    alloc_total += sizeof(*device.val) * crs.nonzeros;
    hipMalloc( (void**)&device.col_ind, sizeof(*device.col_ind) * crs.nonzeros );
    alloc_total += sizeof(*device.col_ind) * crs.nonzeros;
    hipMalloc( (void**)&device.row_ptr, sizeof(*device.row_ptr) * (crs.rows+1) );
    alloc_total += sizeof(*device.row_ptr) * (crs.rows+1);
    hipMalloc( (void**)&device.vector, sizeof(*device.vector) * v.size );
    alloc_total += sizeof(*device.vector) * v.size;
    hipMalloc( (void**)&device.tmp, sizeof(*device.tmp) * crs.nonzeros );
    alloc_total += sizeof(*device.tmp) * crs.nonzeros;
    hipMalloc( (void**)&device.result, sizeof(*device.result) * crs.rows );
    alloc_total += sizeof(*device.result) * crs.rows;
#ifdef WITH_GATHER
    hipMalloc( (void**)&device.gather, sizeof(*device.gather) * crs.nonzeros );
    alloc_total += sizeof(*device.gather) * crs.nonzeros;
#endif
    printf( "GPU memory allocation size: %zd bytes\n", alloc_total );
    
    TIMESTART;
    hipMemcpy( device.val, crs.val, sizeof(*device.val)*crs.nonzeros, hipMemcpyHostToDevice );
    hipMemcpy( device.vector, v.data, sizeof(*device.vector)*v.size, hipMemcpyHostToDevice );
    hipMemcpy( device.col_ind, crs.col_ind, sizeof(*device.col_ind)*crs.nonzeros, hipMemcpyHostToDevice );
    hipMemcpy( device.row_ptr, crs.row_ptr, sizeof(*device.row_ptr)*(crs.rows+1), hipMemcpyHostToDevice );
    TIMEEND("data transfer time (host to device)");

    dim3 blocks(32,32);
    dim3 threads(32,1);
    int loopcount = 20;

#ifdef WITH_GATHER
    TIMESTART;
    gather<<<blocks, threads>>>( device.vector, device.col_ind, device.gather, crs.nonzeros );
    TIMEEND( "gathering data" );
#endif

    TIMESTART;
    for( int loop=0; loop<loopcount; loop++ ) {
        //__global__ void compute( int *val, int *gather, int *tmp, int len ) {
#ifdef WITH_GATHER
        compute<<<blocks, threads>>>( device.val, device.gather, device.tmp, crs.nonzeros );
#else
        compute<<<blocks, threads>>>( device.val, device.col_ind, device.vector, device.tmp, crs.nonzeros );
#endif
        collect<<<blocks, threads>>>( device.row_ptr, device.tmp, device.result, crs.rows );
    }
    TIMEEND( "* COMPUTING TIME *" );

    // transfer result data to host and dump it to stderr
    int *p = (int*)malloc( sizeof(int) * crs.rows );
    hipMemcpy( p, device.result, sizeof(*p)*crs.rows, hipMemcpyDeviceToHost );
    dump_array( stderr, p, crs.rows );

    // clean up
    crs_delete( &crs );
    vec_delete( &v );
    free(p);
    hipFree( device.val );
    hipFree( device.col_ind );
    hipFree( device.row_ptr );
    hipFree( device.vector );
    hipFree( device.tmp );
    hipFree( device.result );

    return 0;
}
